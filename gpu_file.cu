#include <assert.h>
#include <fcntl.h>
#include <stdio.h>
#include <sys/stat.h>
#include <unistd.h>

#include "gpu_file.ch"
#include "ringbuf.ch"
#include "types.ch"
#include "util.ch"

__device__ __constant__ global_file_meta_table_t  * global_file_meta_table;

/* Allocate the global file table */
__host__
void init_gpu_file() {
  global_file_meta_table_t * dev_ptr;
  CUDA_CALL(hipMalloc((void **) &dev_ptr, NUM_BLOCKS * sizeof(file_meta_table_t)));
  CUDA_CALL(hipMemset(dev_ptr, 0, NUM_BLOCKS * sizeof(file_meta_table_t)));
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(global_file_meta_table), &dev_ptr,
                               sizeof(global_file_meta_table_t *)));
}

__device__
static file_t * get_file_from_gpu_fd(gpu_fd fd) {
  file_meta_table_t * file_meta_table =
        (file_meta_table_t *) &(global_file_meta_table[blockIdx.x * MAX_FILES]);
  file_t * cur_file = &(file_meta_table->files[fd]);

  if (!cur_file->in_use) {
    printf("Bad gpu_fd\n");
    return NULL;
  } else {
    return cur_file;
  }
}

__device__
gpu_fd gpu_file_open(char * file_name, permissions_t permissions) {
  // TODO may need to make gpu_file_* into a single thread function
  request_t open_request    = {0};
  open_request.request_type = OPEN_REQUEST;
  open_request.permissions  = permissions;
  gpu_str_cpy(file_name, open_request.file_name, MAX_PATH_SIZE);

  response_t response = {0};
  gpu_enqueue(&open_request, &response);

  printf("debug placeholder %s\n", response.file_data);
  file_meta_table_t * file_meta_table =
        (file_meta_table_t *) &(global_file_meta_table[blockIdx.x * MAX_FILES]);

  /* Fill in slot in file descriptor table */
  for (int i = 0; i < MAX_FILES; i++) {
    if (!file_meta_table->files[i].in_use) {
      file_meta_table->files[i] = {
        .in_use = true,
        .host_fd = response.host_fd,
        .current_size = (size_t) response.file_size,
        .data = response.file_data,
        .permissions = response.permissions,
        .offset = 0,
      };

      return i;
    }
  }

  return FILE_TABLE_FULL;
}

/* Read will return a pointer to memory starting at the 
   read offset. This buffer can also be used to write,
   as the memory in the buffer will be copied back to
   the file upon close. */
__device__
size_t gpu_file_read(gpu_fd fd, size_t size, char ** data_ptr) {
  file_t * cur_file = get_file_from_gpu_fd(fd);
  // TODO check file permissions

  size_t read_size = 0;
  if ((cur_file->offset + size) > cur_file->current_size) {
    read_size = cur_file->current_size - cur_file->offset;
  } else {
    read_size = size;
  }

  *data_ptr = (cur_file->data + cur_file->offset);
  cur_file->offset += read_size;

  return read_size;
}

__device__
off_t gpu_file_seek(gpu_fd fd, off_t offset, int whence) {
  file_t * cur_file = get_file_from_gpu_fd(fd);

  off_t new_offset = -1;
  switch (whence) {
    case SEEK_SET:
      if (offset >= cur_file->current_size) {
        new_offset = offset;
      }
      break;
    case SEEK_CUR:
      // TODO
      break;
    case SEEK_END:
      // TODO
      break;
  }

  cur_file->offset = new_offset;
  return new_offset;
}

__device__
void gpu_file_grow(void) {
  ; 
}

__device__
void gpu_file_close(gpu_fd fd) { 
  file_t * cur_file = get_file_from_gpu_fd(fd);
  request_t close_request   = {0};
  close_request.request_type = CLOSE_REQUEST;
  close_request.host_fd      = cur_file->host_fd;

  response_t response = {0};
  gpu_enqueue(&close_request, &response);


  printf("debug placeholder %s\n", response.file_data);
  // TODO if make this multithread may need to single thread this or
  // something because the file table is per block, not per thread
  /* Free up gpu file descriptor */
  *cur_file = {0};
}

__host__
void handle_gpu_file_open(volatile request_t * request, volatile response_t * ret_response) {
  permissions_t permissions = request->permissions;
  char * file_name = (char *) request->file_name;

  int oflag = 0;//O_CREAT;
  if (permissions == R___)
    oflag |= O_RDONLY;
  else if (permissions == _W__)
    oflag |= O_WRONLY;
  else if (permissions == RW__)
    oflag |= O_RDWR;

  int fd = open(file_name, oflag);
  if (fd == -1)
    perror("handle_gpu_file_open open() failed\n");

  struct stat file_stat;
  int err = fstat(fd, &file_stat);
  if (err == -1)
    perror("handle_gpu_file_open fstat() failed\n");

  off_t file_size = file_stat.st_size;
  // TODO should be able to create files, but cant currently
  assert(file_size > 0); 
 
  char * file_mem = NULL; 
  CUDA_CALL(hipMallocManaged(&file_mem, file_size));
  ssize_t bytes_read = read(fd, file_mem, file_size);
  if (bytes_read != file_size)
    perror("handle_gpu_file_open error reading file\n");

  ret_response->host_fd     = fd;
  ret_response->file_size   = file_size;
  ret_response->permissions = permissions;
  ret_response->file_data   = file_mem;
}

__host__
void handle_gpu_file_grow(volatile request_t * request, volatile response_t * ret_response) {
  // TODO might be best to close the file and flush then grow and reopen
  ;
}

__host__
void handle_gpu_file_close(volatile request_t * request, volatile response_t * ret_response) {
  // TODO probably easier to use ftruncate to grow the open file
  int host_fd = request->host_fd;
  // TODO need ot add fields for actual size
  // original size
  // pointer to memory
  // might want to make request a union of structs
  ;
}
