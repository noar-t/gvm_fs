#include "hip/hip_runtime.h"
#include <stdio.h>

#include "request.ch"
#include "ringbuf.ch"
#include "util.ch"

// TODO add fd array for each process

__host__
ringbuf_t * init_ringbuf() {
  ringbuf_t * ringbuf = NULL;
  CUDA_CALL(hipMallocManaged(&ringbuf, sizeof(ringbuf_t)));

  int dev_id;
  CUDA_CALL(hipGetDevice(&dev_id));
  // TODO required higher cuda version
  //CUDA_CALL(hipMemAdvise(ringbuf, sizeof(ringbuf_t), hipMemAdviseSetAccessedBy, dev_id));
  CUDA_CALL(hipMemset(ringbuf, 0, sizeof(ringbuf_t)));

  ringbuf->cpu_mutex = (cpu_mutex_t *) malloc(sizeof(gpu_mutex_t));
  CUDA_CALL(hipMalloc(&ringbuf->gpu_mutex, sizeof(gpu_mutex_t)));
  CUDA_CALL(hipMemset(ringbuf->gpu_mutex, 0, sizeof(gpu_mutex_t)));

  return ringbuf;
}

__host__
void free_ringbuf(ringbuf_t * ringbuf) {
  CUDA_CALL(hipFree(ringbuf));
}


__host__
bool cpu_dequeue(ringbuf_t * ringbuf, request_t * ret_request) {
  bool success = true;

  CPU_SPINLOCK_LOCK(ringbuf->cpu_mutex);
  unsigned int read_index = ringbuf->read_index;
  if (read_index == ringbuf->write_index)
    return false;

  if (read_index >= (RINGBUF_SIZE-1)) {
    ringbuf->read_index = 0;
    read_index = 0;
  } // HANDLE wrap around at the end
  CPU_SPINLOCK_UNLOCK(ringbuf->cpu_mutex);

  request_t * cur_request = &(ringbuf->requests[ringbuf->read_index]);
 
  while (!cur_request->ready_to_read) {
    *ret_request = *cur_request;
    memset(cur_request, 0, sizeof(request_t));
  }

  __sync_synchronize();
  return success;
}

/* Get a valid write_index
   write into the index
   */
__device__
bool gpu_enqueue(ringbuf_t * ringbuf, request_t * new_request) {
  BEGIN_SINGLE_THREAD;
  GPU_SPINLOCK_LOCK(&ringbuf->gpu_mutex);

  //unsigned int write_index = ringbuf->write_index;
  bool wait = true;
  //while (wait) {
  //  unsigned local_tmp = ringbuf->tmp_counter;

  //  if (local_tmp >= RINGBUF_SIZE) {
  //    unsigned old = atomicCAS(&ringbuf->tmp_counter, local_tmp, 0);
  //    if (old != 0)
  //      continue; /* If atomic failed retry */

  //  } else {
  //    unsigned index = atomicAdd(&ringbuf->tmp_counter, 1);
  //    if (index >= RINGBUF_SIZE) {
  //      continue; /* Bad index value, try again */
  //    }
  //  }

  //  ///* wrap index around */
  //  //if (write_index >= (RINGBUF_SIZE - 1)) {
  //  //  ringbuf->write_index = 0;
  //  //  write_index = 0;
  //  //}

  //  ///* buffer is full */
  //  //if (write_index == ringbuf->read_index && !(write_index == 0 && ringbuf->read_index == 0)) {
  //  //  return false; // TODO come up with more graceful error

  //  //} else { /* take write slot */
  //  //  ringbuf->write_index++;
  //  //}
  //  
  __threadfence_system();
  ringbuf->tmp_counter++;
  printf("block id:%d counter:%d\n", blockIdx.x, ringbuf->tmp_counter);
  //  __threadfence();
  __threadfence_system();

  //  //ringbuf->requests[write_index] = *new_request;
  //  
  //  //ringbuf->requests[write_index].ready_to_read = true;

  //  //__threadfence_system();
  //}
  GPU_SPINLOCK_UNLOCK(&ringbuf->gpu_mutex);
  END_SINGLE_THREAD;
  
  return true;
}

