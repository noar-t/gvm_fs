#include <stdio.h>
#include <inttypes.h>

#include "types.ch"
#include "memory_pool.ch"

#define FREE 0
#define ALLOCATED 1

void * memory_pool;

// false is free, true is allocated
static bool memory_pool_usage_map[NUM_POOL_PAGES];

__host__
void init_memory_pool() {
  CUDA_CALL(hipMallocManaged((void **) memory_pool, MEM_POOL_SIZE));
  memset(memory_pool, 0, MEM_POOL_SIZE);
}

__host__
void toggle_range(int start, int length, bool val) {
  for (int i = start; i <= length; i++) {
    memory_pool_usage_map[i] = val;
  }
}

__host__
int find_free_slot(size_t num_pages_requested) {
  int run_start = 0;
  int current_run = 0; 

  for (int i = 0; i < NUM_POOL_PAGES; i++) {
    if (memory_pool_usage_map[i] == FREE) {
      current_run++;
      if (current_run >= num_pages_requested) {
        toggle_range(run_start, current_run, ALLOCATED);
        return run_start;
      }
    } else {
      run_start = i;
      current_run = 0;
    }
  }

  return -1;
}

__host__
void * allocate_memory(size_t amount) {
  int num_pages_requested = (amount / MIN_ALLOCATION) + ((amount % MIN_ALLOCATION) > 0);
  int index = find_free_slot(num_pages_requested);

  char * ret_ptr = (char *) memory_pool;
  ret_ptr += (index * MIN_ALLOCATION);
  
  return (void *) ret_ptr;
}

__host__
void free(void * ptr, size_t size) {
  // TODO
  return;
}
