#include "hip/hip_runtime.h"
#include <stdio.h>

#include "ringbuf.ch"
#include "util.ch"

__global__
void fill_queue(ringbuf_t * ringbuf) {
  request_t request = { .request_type = test, .placeholder = 77};
  while (!gpu_enqueue(ringbuf, &request)) {;}
}


int main() {
  printf("MAIN\n");
  ringbuf_t * ringbuf = init_ringbuf();

  fill_queue<<<RINGBUF_SIZE, 32>>>(ringbuf);
 // do {
    printf("counter %d\n", ringbuf->tmp_counter);
//  } while (ringbuf->tmp_counter < 100);

  CUDA_CALL(hipDeviceSynchronize());
  __sync_synchronize();
    printf("RINGBUF_SIZE: %d; counter: %d\n", RINGBUF_SIZE, ringbuf->tmp_counter);
  //for (int i = 0; i < RINGBUF_SIZE; i++) {
    //printf("ringbuf[%d] = {.ready_to_read = %d, .request_type = %d, .placeholder = %d}\n",
    //    i,
    //    ringbuf->requests[i].ready_to_read,
    //    ringbuf->requests[i].request_type,
    //    ringbuf->requests[i].placeholder);
  //}

  free_ringbuf(ringbuf);
}



