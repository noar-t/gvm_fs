#include "hip/hip_runtime.h"
#include <stdio.h>

#include "ringbuf.ch"
#include "util.ch"

__global__
void fill_queue(ringbuf_t * ringbuf) {
  request_t request = { .request_type = test, .placeholder = 77};
  while (!gpu_enqueue(ringbuf, &request)) {;}
}


int main() {
  printf("MAIN\n");
  ringbuf_t * ringbuf = init_ringbuf();

  fill_queue<<<RINGBUF_SIZE * 2, 32>>>(ringbuf);

  CUDA_CALL(hipDeviceSynchronize());
  __sync_synchronize();
  for (int i = 0; i < RINGBUF_SIZE; i++) {
    printf("ringbuf[%d] = {.ready_to_read = %d, .request_type = %d, .placeholder = %d}\n",
        i,
        ringbuf->requests[i].ready_to_read,
        ringbuf->requests[i].request_type,
        ringbuf->requests[i].placeholder);
  }

  free_ringbuf(ringbuf);
}



