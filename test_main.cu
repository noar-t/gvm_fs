#include "hip/hip_runtime.h"
#include <stdio.h>

#include "gpu_file.ch"
#include "ringbuf.ch"
#include "types.ch"
#include "util.ch"

#include <unistd.h>

__global__
void fill_queue(void) {
  if (blockIdx.x == 0 && threadIdx.x == 0)
    gpu_file_open("/home/noah/School/gvm_fs/files/testFile.txt", RW__);
}


int main() {
  printf("MAIN\n");
  init_ringbuf();
  printf("DONE init_ringbuf\n");

  fill_queue<<<RINGBUF_SIZE, 32>>>();

  CUDA_CALL(hipDeviceSynchronize());
  sleep(10);

  free_ringbuf();
}



